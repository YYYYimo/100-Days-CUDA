#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>

// Matrix addition kernel - demonstrates 2D indexing
__global__ void matrixAdd2D(float *A, float *B, float *C, int width, int height) {
    // 2D thread indexing
    int col = blockIdx.x * blockDim.x + threadIdx.x;  // x dimension: column
    int row = blockIdx.y * blockDim.y + threadIdx.y;  // y dimension: row
    
    if (col < width && row < height) {
        int idx = row * width + col;  // Convert to 1D index
        C[idx] = A[idx] + B[idx];
        
        // Print index info (only for first few threads)
        if (col < 4 && row < 4) {
            printf("Thread(%d,%d): blockIdx(%d,%d), threadIdx(%d,%d), globalIdx=%d\n",
                   col, row, blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y, idx);
        }
    }
}

// 3D tensor addition kernel - demonstrates 3D indexing
__global__ void tensor3DAdd(float *A, float *B, float *C, int width, int height, int depth) {
    // 3D thread indexing
    int x = blockIdx.x * blockDim.x + threadIdx.x;  // width
    int y = blockIdx.y * blockDim.y + threadIdx.y;  // height  
    int z = blockIdx.z * blockDim.z + threadIdx.z;  // depth
    
    if (x < width && y < height && z < depth) {
        int idx = z * (width * height) + y * width + x;  // 3D to 1D index conversion
        C[idx] = A[idx] + B[idx];
        
        // Print 3D index info
        if (x < 2 && y < 2 && z < 2) {
            printf("3D Thread(%d,%d,%d): blockIdx(%d,%d,%d), threadIdx(%d,%d,%d), globalIdx=%d\n",
                   x, y, z, blockIdx.x, blockIdx.y, blockIdx.z, 
                   threadIdx.x, threadIdx.y, threadIdx.z, idx);
        }
    }
}

// Detailed index analysis kernel
__global__ void indexAnalysis(int *output, int N) {
    // Calculate global thread ID
    int globalThreadId = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (globalThreadId < N) {
        // Store index info to output array
        output[globalThreadId] = globalThreadId;
        
        // Print detailed index info (only for first 16 threads)
        if (globalThreadId < 16) {
            printf("\n=== Thread %d Analysis ===\n", globalThreadId);
            printf("blockIdx.x = %d\n", blockIdx.x);
            printf("blockDim.x = %d\n", blockDim.x);
            printf("threadIdx.x = %d\n", threadIdx.x);
            printf("gridDim.x = %d\n", gridDim.x);
            printf("Global Thread ID = %d * %d + %d = %d\n", 
                   blockIdx.x, blockDim.x, threadIdx.x, globalThreadId);
        }
    }
}

// Warp index demonstration
__global__ void warpAnalysis() {
    int globalId = blockIdx.x * blockDim.x + threadIdx.x;
    int warpId = threadIdx.x / 32;  // warp size is fixed at 32
    int laneId = threadIdx.x % 32;  // position within the warp
    
    if (globalId < 64) {  // Only print for first 64 threads
        printf("Global:%d, Block:%d, Thread:%d, Warp:%d, Lane:%d\n",
               globalId, blockIdx.x, threadIdx.x, warpId, laneId);
    }
}

int main() {
    printf("=== CUDA Indexing System Demo ===\n\n");
    
    // 1. 1D index analysis
    printf("1. 1D Index Analysis:\n");
    int N = 32;
    int *d_output;
    hipMalloc(&d_output, N * sizeof(int));
    
    int threadsPerBlock = 8;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    
    printf("Launch config: %d blocks, %d threads per block\n", blocksPerGrid, threadsPerBlock);
    indexAnalysis<<<blocksPerGrid, threadsPerBlock>>>(d_output, N);
    hipDeviceSynchronize();
    
    // 2. Warp analysis
    printf("\n2. Warp Analysis:\n");
    printf("Block size: 64, 32 threads per warp\n");
    warpAnalysis<<<1, 64>>>();
    hipDeviceSynchronize();
    
    // 3. 2D matrix addition
    printf("\n3. 2D Matrix Indexing:\n");
    int width = 8, height = 6;
    int matSize = width * height * sizeof(float);
    
    float *h_A = (float*)malloc(matSize);
    float *h_B = (float*)malloc(matSize);
    float *h_C = (float*)malloc(matSize);
    float *d_A, *d_B, *d_C;
    
    // Initialize matrices
    for(int i = 0; i < width * height; i++) {
        h_A[i] = 1.0f;
        h_B[i] = 2.0f;
    }
    
    hipMalloc(&d_A, matSize);
    hipMalloc(&d_B, matSize);
    hipMalloc(&d_C, matSize);
    
    hipMemcpy(d_A, h_A, matSize, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, matSize, hipMemcpyHostToDevice);
    
    // 2D grid configuration
    dim3 blockSize2D(4, 3);  // 4x3 = 12 threads per block
    dim3 gridSize2D((width + blockSize2D.x - 1) / blockSize2D.x,
                    (height + blockSize2D.y - 1) / blockSize2D.y);
    
    printf("2D config: Grid(%d,%d), Block(%d,%d)\n", 
           gridSize2D.x, gridSize2D.y, blockSize2D.x, blockSize2D.y);
    
    matrixAdd2D<<<gridSize2D, blockSize2D>>>(d_A, d_B, d_C, width, height);
    hipDeviceSynchronize();
    
    // 4. 3D tensor addition
    printf("\n4. 3D Tensor Indexing:\n");
    int w = 4, h = 3, d = 2;
    int tensorSize = w * h * d * sizeof(float);
    
    float *h_A3D = (float*)malloc(tensorSize);
    float *h_B3D = (float*)malloc(tensorSize);
    float *h_C3D = (float*)malloc(tensorSize);
    float *d_A3D, *d_B3D, *d_C3D;
    
    for(int i = 0; i < w * h * d; i++) {
        h_A3D[i] = 1.0f;
        h_B3D[i] = 2.0f;
    }
    
    hipMalloc(&d_A3D, tensorSize);
    hipMalloc(&d_B3D, tensorSize);
    hipMalloc(&d_C3D, tensorSize);
    
    hipMemcpy(d_A3D, h_A3D, tensorSize, hipMemcpyHostToDevice);
    hipMemcpy(d_B3D, h_B3D, tensorSize, hipMemcpyHostToDevice);
    
    // 3D grid configuration
    dim3 blockSize3D(2, 2, 2);  // 2x2x2 = 8 threads per block
    dim3 gridSize3D((w + blockSize3D.x - 1) / blockSize3D.x,
                    (h + blockSize3D.y - 1) / blockSize3D.y,
                    (d + blockSize3D.z - 1) / blockSize3D.z);
    
    printf("3D config: Grid(%d,%d,%d), Block(%d,%d,%d)\n", 
           gridSize3D.x, gridSize3D.y, gridSize3D.z,
           blockSize3D.x, blockSize3D.y, blockSize3D.z);
    
    tensor3DAdd<<<gridSize3D, blockSize3D>>>(d_A3D, d_B3D, d_C3D, w, h, d);
    hipDeviceSynchronize();
    
    // Free memory
    hipFree(d_output);
    hipFree(d_A); hipFree(d_B); hipFree(d_C);
    hipFree(d_A3D); hipFree(d_B3D); hipFree(d_C3D);
    free(h_A); free(h_B); free(h_C);
    free(h_A3D); free(h_B3D); free(h_C3D);
    
    printf("\n=== Indexing System Demo Finished ===\n");
    return 0;
}