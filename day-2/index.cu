#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>

// ����ӷ��ں� - չʾ2D����
__global__ void matrixAdd2D(float *A, float *B, float *C, int width, int height) {
    // 2D�߳�����
    int col = blockIdx.x * blockDim.x + threadIdx.x;  // xά�ȣ���
    int row = blockIdx.y * blockDim.y + threadIdx.y;  // yά�ȣ���
    
    if (col < width && row < height) {
        int idx = row * width + col;  // ת��Ϊ1D����
        C[idx] = A[idx] + B[idx];
        
        // ��ӡ������Ϣ����ǰ�����̣߳�
        if (col < 4 && row < 4) {
            printf("Thread(%d,%d): blockIdx(%d,%d), threadIdx(%d,%d), globalIdx=%d\n",
                   col, row, blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y, idx);
        }
    }
}

// 3D�����ӷ��ں� - չʾ3D����
__global__ void tensor3DAdd(float *A, float *B, float *C, int width, int height, int depth) {
    // 3D�߳�����
    int x = blockIdx.x * blockDim.x + threadIdx.x;  // ���
    int y = blockIdx.y * blockDim.y + threadIdx.y;  // �߶�  
    int z = blockIdx.z * blockDim.z + threadIdx.z;  // ���
    
    if (x < width && y < height && z < depth) {
        int idx = z * (width * height) + y * width + x;  // 3D��1D����ת��
        C[idx] = A[idx] + B[idx];
        
        // ��ӡ3D������Ϣ
        if (x < 2 && y < 2 && z < 2) {
            printf("3D Thread(%d,%d,%d): blockIdx(%d,%d,%d), threadIdx(%d,%d,%d), globalIdx=%d\n",
                   x, y, z, blockIdx.x, blockIdx.y, blockIdx.z, 
                   threadIdx.x, threadIdx.y, threadIdx.z, idx);
        }
    }
}

// ��ϸ���������ں�
__global__ void indexAnalysis(int *output, int N) {
    // ����ȫ���߳�ID
    int globalThreadId = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (globalThreadId < N) {
        // ��������Ϣ�洢���������
        output[globalThreadId] = globalThreadId;
        
        // ��ӡ��ϸ������Ϣ����ǰ16���̣߳�
        if (globalThreadId < 16) {
            printf("\n=== Thread %d Analysis ===\n", globalThreadId);
            printf("blockIdx.x = %d\n", blockIdx.x);
            printf("blockDim.x = %d\n", blockDim.x);
            printf("threadIdx.x = %d\n", threadIdx.x);
            printf("gridDim.x = %d\n", gridDim.x);
            printf("Global Thread ID = %d * %d + %d = %d\n", 
                   blockIdx.x, blockDim.x, threadIdx.x, globalThreadId);
        }
    }
}

// չʾwarp����
__global__ void warpAnalysis() {
    int globalId = blockIdx.x * blockDim.x + threadIdx.x;
    int warpId = threadIdx.x / 32;  // warp��С�̶�Ϊ32
    int laneId = threadIdx.x % 32;  // ��warp�ڵ�λ��
    
    if (globalId < 64) {  // ֻ��ӡǰ64���߳�
        printf("Global:%d, Block:%d, Thread:%d, Warp:%d, Lane:%d\n",
               globalId, blockIdx.x, threadIdx.x, warpId, laneId);
    }
}

int main() {
    printf("=== CUDA ����ϵͳ��� ===\n\n");
    
    // 1. һά��������
    printf("1. һά��������:\n");
    int N = 32;
    int *d_output;
    hipMalloc(&d_output, N * sizeof(int));
    
    int threadsPerBlock = 8;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    
    printf("��������: %d blocks, %d threads per block\n", blocksPerGrid, threadsPerBlock);
    indexAnalysis<<<blocksPerGrid, threadsPerBlock>>>(d_output, N);
    hipDeviceSynchronize();
    
    // 2. Warp����
    printf("\n2. Warp ����:\n");
    printf("Block size: 64, ÿ��warp 32���߳�\n");
    warpAnalysis<<<1, 64>>>();
    hipDeviceSynchronize();
    
    // 3. ��ά����ӷ�
    printf("\n3. ��ά��������:\n");
    int width = 8, height = 6;
    int matSize = width * height * sizeof(float);
    
    float *h_A = (float*)malloc(matSize);
    float *h_B = (float*)malloc(matSize);
    float *h_C = (float*)malloc(matSize);
    float *d_A, *d_B, *d_C;
    
    // ��ʼ������
    for(int i = 0; i < width * height; i++) {
        h_A[i] = 1.0f;
        h_B[i] = 2.0f;
    }
    
    hipMalloc(&d_A, matSize);
    hipMalloc(&d_B, matSize);
    hipMalloc(&d_C, matSize);
    
    hipMemcpy(d_A, h_A, matSize, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, matSize, hipMemcpyHostToDevice);
    
    // 2D��������
    dim3 blockSize2D(4, 3);  // 4x3 = 12���߳�ÿ��
    dim3 gridSize2D((width + blockSize2D.x - 1) / blockSize2D.x,
                    (height + blockSize2D.y - 1) / blockSize2D.y);
    
    printf("2D����: Grid(%d,%d), Block(%d,%d)\n", 
           gridSize2D.x, gridSize2D.y, blockSize2D.x, blockSize2D.y);
    
    matrixAdd2D<<<gridSize2D, blockSize2D>>>(d_A, d_B, d_C, width, height);
    hipDeviceSynchronize();
    
    // 4. ��ά�����ӷ�
    printf("\n4. ��ά��������:\n");
    int w = 4, h = 3, d = 2;
    int tensorSize = w * h * d * sizeof(float);
    
    float *h_A3D = (float*)malloc(tensorSize);
    float *h_B3D = (float*)malloc(tensorSize);
    float *h_C3D = (float*)malloc(tensorSize);
    float *d_A3D, *d_B3D, *d_C3D;
    
    for(int i = 0; i < w * h * d; i++) {
        h_A3D[i] = 1.0f;
        h_B3D[i] = 2.0f;
    }
    
    hipMalloc(&d_A3D, tensorSize);
    hipMalloc(&d_B3D, tensorSize);
    hipMalloc(&d_C3D, tensorSize);
    
    hipMemcpy(d_A3D, h_A3D, tensorSize, hipMemcpyHostToDevice);
    hipMemcpy(d_B3D, h_B3D, tensorSize, hipMemcpyHostToDevice);
    
    // 3D��������
    dim3 blockSize3D(2, 2, 2);  // 2x2x2 = 8���߳�ÿ��
    dim3 gridSize3D((w + blockSize3D.x - 1) / blockSize3D.x,
                    (h + blockSize3D.y - 1) / blockSize3D.y,
                    (d + blockSize3D.z - 1) / blockSize3D.z);
    
    printf("3D����: Grid(%d,%d,%d), Block(%d,%d,%d)\n", 
           gridSize3D.x, gridSize3D.y, gridSize3D.z,
           blockSize3D.x, blockSize3D.y, blockSize3D.z);
    
    tensor3DAdd<<<gridSize3D, blockSize3D>>>(d_A3D, d_B3D, d_C3D, w, h, d);
    hipDeviceSynchronize();
    
    // �����ڴ�
    hipFree(d_output);
    hipFree(d_A); hipFree(d_B); hipFree(d_C);
    hipFree(d_A3D); hipFree(d_B3D); hipFree(d_C3D);
    free(h_A); free(h_B); free(h_C);
    free(h_A3D); free(h_B3D); free(h_C3D);
    
    printf("\n=== ����ϵͳ��ʾ��� ===\n");
    return 0;
}