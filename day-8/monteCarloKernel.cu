#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#define THREADS_PER_BLOCK 256

// CUDA kernel for Monte Carlo integration
__global__ void monteCarloKernel(float *results, int samples, unsigned int seed) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int totalThreads = gridDim.x * blockDim.x;

    hiprandState state;
    hiprand_init(seed, idx, 0, &state);

    float sum = 0.0f;
    for (int i = idx; i < samples; i += totalThreads) {
        float x = hiprand_uniform(&state); // x in (0,1]
        float y = x * x; // Integrate y = x^2
        sum += y;
    }
    results[idx] = sum;
}

int main() {
    int samples = 10000000;
    int blocks = 128;
    int threads = THREADS_PER_BLOCK;
    int totalThreads = blocks * threads;

    float *d_results, *h_results;
    h_results = (float*)malloc(totalThreads * sizeof(float));
    hipMalloc(&d_results, totalThreads * sizeof(float));

    monteCarloKernel<<<blocks, threads>>>(d_results, samples, time(NULL));
    hipDeviceSynchronize();

    hipMemcpy(h_results, d_results, totalThreads * sizeof(float), hipMemcpyDeviceToHost);

    double total = 0.0;
    for (int i = 0; i < totalThreads; ++i) {
        total += h_results[i];
    }
    double estimate = total / samples; // [0,1]���䣬������Ǿ�ֵ

    printf("Monte Carlo estimate of integral of x^2 over [0,1]: %.6f\n", estimate);
    printf("Theoretical value: 1/3 = %.6f\n", 1.0/3.0);

    hipFree(d_results);
    free(h_results);
    return 0;
}